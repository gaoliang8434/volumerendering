#include "hip/hip_runtime.h"
#include <float.h>
#include <hip/hip_vector_types.h>
#include "cuda/cuFieldStructs.cuh"

#define HIP_PI_F (3.141592654f)

inline __device__ float4 floor(float4 a){
   return ( (float4){floor(a.x), floor(a.y), floor(a.z), floor(a.w)} );
}

__device__ float4 unitvector(float4 v)
{
   float vmag = sqrt(v.x*v.x + v.y*v.y + v.z*v.z);
   return v/vmag;
}

__device__ bool isInside(int i, int j, int k, int nx, int ny, int nz )
{
   return (
      (i > -1) && (i < nx) &&
      (j > -1) && (j < ny) &&
      (k > -1) && (k < nz)
   );
}

__device__ float4 cameraTransform(float4 X, cu_DSM& dsm)
{
   X = X - dsm.camera_position;
   float z = dot(X, dsm.camera_view);
   if( z < 0 )
      return make_float4(0.0f);

   float x = dot(X, dsm.camera_right) / z;
   float y = dot(X, dsm.camera_up) / z;
   x = (x / dsm.htanfov + 1.0) / 2.0;
   y = (y / dsm.vtanfov + 1.0) / 2.0;
   z = (length(X) - dsm.near) / (dsm.far - dsm.near);
   return make_float4(x, y, z, 0.0f);
}

template <typename T>
__device__ T get(
      const T* data, const long* map, const long* mapOfMap,
      int i, int j, int k,
      int nx, int ny, int nz,
      int nnx, int nny, int partitionSize,
      T defVal
)
{
   if(!isInside( i, j, k, nx, ny, nz )){ return (defVal); }

   int ii = i / (partitionSize * partitionSize);
   int jj = j / (partitionSize * partitionSize);
   int kk = k / (partitionSize * partitionSize);
   int mapIndex = ii + nnx * (jj + nny * kk);
   if(mapOfMap[mapIndex] == -1)
   {
      return (defVal);
   }
   else
   {
      long offset = mapOfMap[mapIndex];
      ii = (i / partitionSize) % partitionSize;
      jj = (j / partitionSize) % partitionSize;
      kk = (k / partitionSize) % partitionSize;
      mapIndex = ii + partitionSize * (jj + partitionSize * kk);
      if(map[mapIndex + offset] == -1)
      {
         return (defVal);
      }
      else
      {
         offset = map[mapIndex + offset];
         ii = i % partitionSize;
         jj = j % partitionSize;
         kk = k % partitionSize;
         mapIndex = ii + partitionSize * (jj + partitionSize * kk);
         return data[mapIndex + offset];
      }
   }
}

__device__ float valueAtLinearInterpolation( float4 X, cu_ScalarField& field )
{
   X = X - field.LLC;

   // Get integer coordinates
   float4 a = make_float4( X.x/field.Res.x, X.y/field.Res.y, X.z/field.Res.z, 0.0f );

   if(
      a.x < 0 || a.x >= field.nx ||
      a.y < 0 || a.y >= field.ny ||
      a.z < 0 || a.z >= field.nz
   )
   { return field.defVal; }

   // Take the floor of each
   int4 b;
   b.x = (int)(floor(a.x));
   b.y = (int)(floor(a.y));
   b.z = (int)(floor(a.z));
   b.w = 0;

   // Get the weights
   float4 s = a - floor(a);
   float4 t = make_float4( 1.f - s.x, 1.f - s.y, 1.f - s.z, 0.0f );

   float psize = field.partitionSize;
   return
      get(field.data, field.map, field.mapOfMap, b.x,   b.y,   b.z,   field.nx, field.ny, field.nz, field.nnx, field.nny, psize, field.defVal) * (t.x * t.y * t.z) +
      get(field.data, field.map, field.mapOfMap, b.x,   b.y,   b.z+1, field.nx, field.ny, field.nz, field.nnx, field.nny, psize, field.defVal) * (t.x * t.y * s.z) +
      get(field.data, field.map, field.mapOfMap, b.x,   b.y+1, b.z,   field.nx, field.ny, field.nz, field.nnx, field.nny, psize, field.defVal) * (t.x * s.y * t.z) +
      get(field.data, field.map, field.mapOfMap, b.x,   b.y+1, b.z+1, field.nx, field.ny, field.nz, field.nnx, field.nny, psize, field.defVal) * (t.x * s.y * s.z) +
      get(field.data, field.map, field.mapOfMap, b.x+1, b.y,   b.z,   field.nx, field.ny, field.nz, field.nnx, field.nny, psize, field.defVal) * (s.x * t.y * t.z) +
      get(field.data, field.map, field.mapOfMap, b.x+1, b.y,   b.z+1, field.nx, field.ny, field.nz, field.nnx, field.nny, psize, field.defVal) * (s.x * t.y * s.z) +
      get(field.data, field.map, field.mapOfMap, b.x+1, b.y+1, b.z,   field.nx, field.ny, field.nz, field.nnx, field.nny, psize, field.defVal) * (s.x * s.y * t.z) +
      get(field.data, field.map, field.mapOfMap, b.x+1, b.y+1, b.z+1, field.nx, field.ny, field.nz, field.nnx, field.nny, psize, field.defVal) * (s.x * s.y * s.z);
}

__device__ float dsmValueAtLinearInterpolation( float4 X, cu_DSM& dsm )
{
   X = cameraTransform(X, dsm);

   if(
      X.x < 0 || X.x > 1.0 ||
      X.y < 0 || X.y > 1.0 ||
      X.z < 0 || X.z > 1.0
   )
   { return dsm.field.defVal; }

   // Get integer coordinates
   float4 a = make_float4( X.x * dsm.field.nx, X.y * dsm.field.ny, X.z * dsm.field.nz, 0.0f );

   // Take the floor of each
   int4 b;
   b.x = (int)(floor(a.x));
   b.y = (int)(floor(a.y));
   b.z = (int)(floor(a.z));
   b.w = 0;

   // Get the weights
   float4 s = a - floor(a);
   float4 t = make_float4( 1.f - s.x, 1.f - s.y, 1.f - s.z, 0.0f );

   float psize = dsm.field.partitionSize;
   return
      get(dsm.field.data, dsm.field.map, dsm.field.mapOfMap, b.x,   b.y,   b.z,   dsm.field.nx, dsm.field.ny, dsm.field.nz, dsm.field.nnx, dsm.field.nny, psize, dsm.field.defVal) * (t.x * t.y * t.z) +
      get(dsm.field.data, dsm.field.map, dsm.field.mapOfMap, b.x,   b.y,   b.z+1, dsm.field.nx, dsm.field.ny, dsm.field.nz, dsm.field.nnx, dsm.field.nny, psize, dsm.field.defVal) * (t.x * t.y * s.z) +
      get(dsm.field.data, dsm.field.map, dsm.field.mapOfMap, b.x,   b.y+1, b.z,   dsm.field.nx, dsm.field.ny, dsm.field.nz, dsm.field.nnx, dsm.field.nny, psize, dsm.field.defVal) * (t.x * s.y * t.z) +
      get(dsm.field.data, dsm.field.map, dsm.field.mapOfMap, b.x,   b.y+1, b.z+1, dsm.field.nx, dsm.field.ny, dsm.field.nz, dsm.field.nnx, dsm.field.nny, psize, dsm.field.defVal) * (t.x * s.y * s.z) +
      get(dsm.field.data, dsm.field.map, dsm.field.mapOfMap, b.x+1, b.y,   b.z,   dsm.field.nx, dsm.field.ny, dsm.field.nz, dsm.field.nnx, dsm.field.nny, psize, dsm.field.defVal) * (s.x * t.y * t.z) +
      get(dsm.field.data, dsm.field.map, dsm.field.mapOfMap, b.x+1, b.y,   b.z+1, dsm.field.nx, dsm.field.ny, dsm.field.nz, dsm.field.nnx, dsm.field.nny, psize, dsm.field.defVal) * (s.x * t.y * s.z) +
      get(dsm.field.data, dsm.field.map, dsm.field.mapOfMap, b.x+1, b.y+1, b.z,   dsm.field.nx, dsm.field.ny, dsm.field.nz, dsm.field.nnx, dsm.field.nny, psize, dsm.field.defVal) * (s.x * s.y * t.z) +
      get(dsm.field.data, dsm.field.map, dsm.field.mapOfMap, b.x+1, b.y+1, b.z+1, dsm.field.nx, dsm.field.ny, dsm.field.nz, dsm.field.nnx, dsm.field.nny, psize, dsm.field.defVal) * (s.x * s.y * s.z);
}

__device__ float4 valueAtLinearInterpolation4( float4 X, cu_ColorField& field )
{
   X = X - field.LLC;

   // Get integer coordinates
   float4 a = make_float4( X.x/field.Res.x, X.y/field.Res.y, X.z/field.Res.z, 0.0f );

   if(
      a.x < 0 || a.x >= field.nx ||
      a.y < 0 || a.y >= field.ny ||
      a.z < 0 || a.z >= field.nz
   )
   { return field.defVal; }

   // Take the floor of each
   int4 b;
   b.x = (int)(floor(a.x));
   b.y = (int)(floor(a.y));
   b.z = (int)(floor(a.z));
   b.w = 0;

   // Get the weights
   float4 s = a - floor(a);
   float4 t = make_float4( 1.f - s.x, 1.f - s.y, 1.f - s.z, 0.0f );

   float psize = field.partitionSize;
   return
      get(field.data, field.map, field.mapOfMap, b.x,   b.y,   b.z,   field.nx, field.ny, field.nz, field.nnx, field.nny, psize, field.defVal) * (t.x * t.y * t.z) +
      get(field.data, field.map, field.mapOfMap, b.x,   b.y,   b.z+1, field.nx, field.ny, field.nz, field.nnx, field.nny, psize, field.defVal) * (t.x * t.y * s.z) +
      get(field.data, field.map, field.mapOfMap, b.x,   b.y+1, b.z,   field.nx, field.ny, field.nz, field.nnx, field.nny, psize, field.defVal) * (t.x * s.y * t.z) +
      get(field.data, field.map, field.mapOfMap, b.x,   b.y+1, b.z+1, field.nx, field.ny, field.nz, field.nnx, field.nny, psize, field.defVal) * (t.x * s.y * s.z) +
      get(field.data, field.map, field.mapOfMap, b.x+1, b.y,   b.z,   field.nx, field.ny, field.nz, field.nnx, field.nny, psize, field.defVal) * (s.x * t.y * t.z) +
      get(field.data, field.map, field.mapOfMap, b.x+1, b.y,   b.z+1, field.nx, field.ny, field.nz, field.nnx, field.nny, psize, field.defVal) * (s.x * t.y * s.z) +
      get(field.data, field.map, field.mapOfMap, b.x+1, b.y+1, b.z,   field.nx, field.ny, field.nz, field.nnx, field.nny, psize, field.defVal) * (s.x * s.y * t.z) +
      get(field.data, field.map, field.mapOfMap, b.x+1, b.y+1, b.z+1, field.nx, field.ny, field.nz, field.nnx, field.nny, psize, field.defVal) * (s.x * s.y * s.z);
}

// Like valueAtLinearInterpolation4, but with periodic boundary conditions (maybe)
__device__ float4 valueAtLinearInterpolation4Periodic( float4 X, cu_ColorField& field )
{
   X = X - field.LLC;

   // Get integer coordinates
   float4 a = make_float4( X.x/field.Res.x, X.y/field.Res.y, X.z/field.Res.z, 0.0f );

   // Periodic boundary
   a.x += (a.x < 0.0f) ? field.nx : (a.x > field.nx) ? -field.nx : 0.0f;
   a.y += (a.y < 0.0f) ? field.ny : (a.y > field.ny) ? -field.ny : 0.0f;
   a.z += (a.z < 0.0f) ? field.nz : (a.z > field.nz) ? -field.nz : 0.0f;

   // Take the floor of each
   int4 b;
   b.x = (int)(floor(a.x));
   b.y = (int)(floor(a.y));
   b.z = (int)(floor(a.z));
   b.w = 0;

   // Get the weights
   float4 s = a - floor(a);
   float4 t = make_float4( 1.f - s.x, 1.f - s.y, 1.f - s.z, 0.0f );

   float psize = field.partitionSize;
   return
      get(field.data, field.map, field.mapOfMap, b.x,   b.y,   b.z,   field.nx, field.ny, field.nz, field.nnx, field.nny, psize, field.defVal) * (t.x * t.y * t.z) +
      get(field.data, field.map, field.mapOfMap, b.x,   b.y,   b.z+1, field.nx, field.ny, field.nz, field.nnx, field.nny, psize, field.defVal) * (t.x * t.y * s.z) +
      get(field.data, field.map, field.mapOfMap, b.x,   b.y+1, b.z,   field.nx, field.ny, field.nz, field.nnx, field.nny, psize, field.defVal) * (t.x * s.y * t.z) +
      get(field.data, field.map, field.mapOfMap, b.x,   b.y+1, b.z+1, field.nx, field.ny, field.nz, field.nnx, field.nny, psize, field.defVal) * (t.x * s.y * s.z) +
      get(field.data, field.map, field.mapOfMap, b.x+1, b.y,   b.z,   field.nx, field.ny, field.nz, field.nnx, field.nny, psize, field.defVal) * (s.x * t.y * t.z) +
      get(field.data, field.map, field.mapOfMap, b.x+1, b.y,   b.z+1, field.nx, field.ny, field.nz, field.nnx, field.nny, psize, field.defVal) * (s.x * t.y * s.z) +
      get(field.data, field.map, field.mapOfMap, b.x+1, b.y+1, b.z,   field.nx, field.ny, field.nz, field.nnx, field.nny, psize, field.defVal) * (s.x * s.y * t.z) +
      get(field.data, field.map, field.mapOfMap, b.x+1, b.y+1, b.z+1, field.nx, field.ny, field.nz, field.nnx, field.nny, psize, field.defVal) * (s.x * s.y * s.z);
}

__device__ float Theta( float4 nhat, float4 nhatp )
{
   return acos( dot(nhat, nhatp) );
}

__device__ float Ftau( float theta, float K, float tau, float taup )
{
   return( (theta + 2.f * HIP_PI_F * K) * (taup / tau) );
}

__device__ float M( float theta, float K, float tau )
{
   return( (theta + (2.f * HIP_PI_F * K)) / (sqrt(2.0f) * tau) );
}

__device__ float N( float mu )
{
   return( sqrt(HIP_PI_F * 0.5f * mu) / (1.f - exp(-2.f / mu)) );
}

__device__ float Delmo( float mu, float M, float N, float R, float tau, float taup )
{
   float lhs = mu * N / R;
   float rhs = sin(M * taup) * sin( M * (tau - taup) );
   rhs /= ( M * sin(M * tau) );
   return ( lhs * rhs );
}

__device__ float Deloo( float mu, float N, float R, float tau, float taup )
{
   return ( (mu * N / R) * (taup * (tau - taup) / tau) );
}

__device__ float4 Bo( float4 nhatp, float4 rhat, float f )
{
   return( nhatp * cos(f) + rhat * sin(f) );
}

__device__ float4 DXDS( float4 nhat, float4 nhatp, float4 rhat, float tau, float taup, float theta, float K, float mu, float n, float R )
{
//   float theta = Theta(nhat, nhatp);
   float f = Ftau(theta, K, tau, taup);
   float4 bnaught = Bo(nhatp, rhat, f);
//   float n = N(mu);
   float m = M(theta, K, tau);

   float delmo = Delmo(mu, m, n, R, tau, taup);
   float deloo = Deloo(mu, n, R, tau, taup);

   return( bnaught * (1.f + (0.5f * (delmo + deloo))) );
}

__device__ float getR( const float* lookup, int size, float eta )
{
   int minv = 0;
   int maxv = size - 1;

   while( minv <= maxv )
   {
      int ndx = (minv + maxv) / 2;
      if( lookup[ndx] == eta ){
         float R = (float)ndx / (float)(size - 1);
         return ( R );
      }
      else if( lookup[ndx] > eta ){
         minv = ndx + 1;
      }
      else{
         maxv = ndx - 1;
      }
   }

   if( lookup[minv] == eta )
      return ( (float)minv / (float)(size - 1) );
   else if( lookup[maxv] == eta )
      return ( (float)maxv / (float)(size - 1) );
   else if( lookup[minv] > eta && lookup[maxv] > eta )
   {
      while( lookup[minv] > eta && lookup[maxv] > eta )
      {
         minv += 1;
         maxv += 1;
      }
   }
   else if( lookup[minv] < eta && lookup[maxv] < eta )
   {
      while( lookup[minv] < eta && lookup[maxv] < eta )
      {
         minv -= 1;
         maxv -= 1;
      }
   }

   float Rn = (float)minv / (float)(size - 1);
   float Rnm = (float)maxv / (float)(size - 1);
   float peta = (eta - lookup[minv]) / (lookup[maxv] - lookup[minv]);
   return ( peta * (Rn - Rnm) + Rnm );
}

__device__ float getC( const float* clookup, int wsize, float wdtau, float tau )
{
   int index = (int)(floor(tau / wdtau));
   return ( clookup[index] );
}

__device__ float evalPhaseFunction(cuPhaseFunction& phaseF, float theta)
{
   float ret = 0.0f;
   if( phaseF.type == cuPhaseFunction::PhaseFunctionType::HENYEYGREENSTEIN )
   {
      float costheta = cos(theta);
      float denom = 1.0 + phaseF.g0 * phaseF.g0 - 2.0 * phaseF.g0 * costheta;
      ret = (1.0 - phaseF.g0 * phaseF.g0)/(4.0*3.14159265);
      ret /= denom;
   } else if( phaseF.type == cuPhaseFunction::PhaseFunctionType::DOUBLE_HENYEYGREENSTEIN ) {
      float costheta = cos(theta);
      float denom = 1.0 + phaseF.g0 * phaseF.g0 - 2.0 * phaseF.g0 * costheta;
      float pf0 = (1.0 - phaseF.g0 * phaseF.g0)/(4.0*3.14159265);
      pf0 /= denom;

      denom = 1.0 + phaseF.g1 * phaseF.g1 - 2.0 * phaseF.g1 * costheta;
      float pf1 = (1.0 - phaseF.g1 * phaseF.g1)/(4.0*3.14159265);
      pf1 /= denom;

      ret = phaseF.mix * pf0 + (1.0 - phaseF.mix) * pf1;
   } else if( phaseF.type == cuPhaseFunction::PhaseFunctionType::FOURNIERFORAND ) {
      double s = sin( theta/2.0 );
      if( fabs(theta) < 0.00001 ){ s = sin(0.000001/2.0); }
      double s2 = s*s;
      double delta = phaseF.delta180*s2;
      double deltanupower = pow( (double)delta, (double)phaseF.nu );
      double c = cos(theta);
      if( fabs(theta) < 0.00001 ){ c = cos(0.000001); }
      double c2 = c*c;
      double p1 = phaseF.p1factor * ( 3.0*c2 - 1.0 );
      double p0 = phaseF.nu*(1.0-delta) - (1.0-deltanupower) + ( delta*(1.0-deltanupower) - phaseF.nu*(1.0-delta) )/s2;
      p0 /= 4.0*3.14159265*(1.0-delta)*(1.0-delta)*deltanupower;
      float value = p0 + p1;

      ret = value;
   } else {
      ret = phaseF.val;
   }

   return ret;
}

__device__ float4 gatherLight(float4 X, float4 D, float k, cu_DSM *dsmFields, int nbLights, cuPhaseFunction& phaseF, const float4 *lightColor, const float4 *lightPosition)
{
   float4 accum = make_float4(0.0f);
   for(int i = 0; i < nbLights; ++i)
   {
      float dsm = dsmValueAtLinearInterpolation(X, dsmFields[i]);
      float4 scatterDirection = lightPosition[i] - X;
      float cosd = dot(unitvector(scatterDirection), unitvector(D));
      float theta = acos(cosd);

      float pf = evalPhaseFunction(phaseF, theta);
      accum += pf * lightColor[i] * exp( -dsm * k );
   }
   return accum;
}

__global__ void kernel_ssRayMarchAccumulation(
   cu_ScalarField densityField, cu_DSM* dsmFields, cu_ColorField colorField,
   cu_ScalarField ambientDensityField, cu_ColorField ambientColorField,

   const float4* lightColor, const float4* lightPosition, int nbLights,

   const float4* startPosition, const float4* startDirection, int nbRays,
   float scatterCoefficient, float ds, float maxPathlength, float clampv,
   cuPhaseFunction phaseFunction,

   float4* Cd
)
{
   uint item = blockDim.x * blockIdx.x + threadIdx.x;
   if( item >= nbRays ) return;

   float4 accum = make_float4(0.0f);

   float T = 1;
   float s = 0;
   float density, ambDensity, totalDensity, dT;
   float4 color, amb, light;

   float4 X;
   float4 Y = startPosition[item];
   float4 D = startDirection[item];

   float4 boxdem;
   boxdem.x = densityField.Res.x * (float)(densityField.nx);
   boxdem.y = densityField.Res.y * (float)(densityField.ny);
   boxdem.z = densityField.Res.z * (float)(densityField.nz);
   boxdem.w = 0.f;  

   float4 boxmax = densityField.LLC + boxdem;

   float t1, t2, tnear, tfar;
   float4 hits;

   tnear = -FLT_MAX;
   tfar = FLT_MAX;

   /* AABB Test */

   // x-axis
   if( D.x != 0.f || (D.x == 0.f && Y.x > densityField.LLC.x && Y.x < boxmax.x) ){
      t1 = (densityField.LLC.x - Y.x) / D.x;
      t2 = (boxmax.x - Y.x) / D.x;
      
      if( t1 > t2 )
      {
         float temp = t1;
         t1 = t2;
         t2 = temp;
      }
      if( t1 > tnear )
         tnear = t1;
      if( t2 < tfar )
         tfar = t2;
      hits.x = tnear;
   }

   tnear = -FLT_MAX;
   tfar = FLT_MAX;

   // y-axis
   if( D.y != 0.f || (D.y == 0.f && Y.y > densityField.LLC.y && Y.y < boxmax.y) ){
      t1 = (densityField.LLC.y - Y.y) / D.y;
      t2 = (boxmax.y - Y.y) / D.y;
      
      if( t1 > t2 )
      {
         float temp = t1;
         t1 = t2;
         t2 = temp;
      }
      if( t1 > tnear )
         tnear = t1;
      if( t2 < tfar )
         tfar = t2;
      hits.y = tnear;
   }

   tnear = -FLT_MAX;
   tfar = FLT_MAX;

   // z-axis
   if( D.z != 0.f || (D.z == 0.f && Y.z > densityField.LLC.z && Y.z < boxmax.z) ){
      t1 = (densityField.LLC.z - Y.z) / D.z;
      t2 = (boxmax.z - Y.z) / D.z;
      if( t1 > t2 )
      {
         float temp = t1;
         t1 = t2;
         t2 = temp;
      }
      if( t1 > tnear )
         tnear = t1;
      if( t2 < tfar )
         tfar = t2;
      hits.z = tnear;
   }

   tnear = hits.x;
   if( tnear < hits.y )
      tnear = hits.y;
   if( tnear < hits.z )
      tnear = hits.z;

   // tnear = 0;
   // tfar = maxPathlength;
   X = Y + tnear * D;
   s = tnear;
   maxPathlength = tfar - tnear;
   while( s <= maxPathlength && T > 1.0e-6 )
   {
      density = valueAtLinearInterpolation(X, densityField);

      // P = X - ambientDensityField.LLC;
      ambDensity = valueAtLinearInterpolation(X, ambientDensityField);

      density = clamp(density / clampv, 0.0f, 1.0f);
      ambDensity = clamp(ambDensity / clampv, 0.0f, 1.0f);
      if( density + ambDensity > 0 )
      {
         totalDensity = density + ambDensity;

         // P = X - colorField.LLC;
         color = valueAtLinearInterpolation4(X, colorField);

         // P = X - ambientColorField.LLC;
         amb = valueAtLinearInterpolation4(X, ambientColorField);

         light = gatherLight(X, D, scatterCoefficient, dsmFields, nbLights, phaseFunction, lightColor, lightPosition);

         dT = exp( -scatterCoefficient * ds * totalDensity );
         color = (amb * ambDensity + color * light * density) / totalDensity;
         color = color * T * (1.0 - dT);

         accum += color;
         T *= dT;
      }
      s += ds;
      X += D * ds;

   }

   accum.w = 1.0 - T;
   Cd[item] = accum;
}

__global__ void kernel_ssWarpRayMarchAccumulation(
   cu_ScalarField densityField, cu_DSM* dsmFields, cu_ColorField warpField, cu_ColorField colorField,
   cu_ScalarField ambientDensityField, cu_ColorField ambientColorField,

   const float4* lightColor, const float4* lightPosition, int nbLights,

   const float4* startPosition, const float4* startDirection, int nbRays, int nbWarps,
   float scatterCoefficient, float ds, float maxPathlength, float clampv,
   cuPhaseFunction phaseFunction,

   float4* Cd
)
{
   uint item = blockDim.x * blockIdx.x + threadIdx.x;
   if( item >= nbRays ) return;
   {
      float4 accum = make_float4(0.0f);
      float T = 1;
      float s = 0;
      float4 X = startPosition[item];
      float4 D = startDirection[item];
      float density, ambDensity, totalDensity, dT;
      float4 color, amb, light, W, dW;
      int iw;
      while( s < maxPathlength && T > 1.0e-6 )
      {
         W = X - warpField.LLC;
         iw = 0;
         while( iw < nbWarps )
         {
            dW = valueAtLinearInterpolation4Periodic(W, warpField);
            W = W - dW;
            iw++;
         }
         W = W + warpField.LLC;
         // P = W - densityField.LLC;
         density = valueAtLinearInterpolation(W, densityField);

         // P = W - ambientDensityField.LLC;
         ambDensity = valueAtLinearInterpolation(W, ambientDensityField);

         density = clamp(density / clampv, 0.0f, 1.0f);
         ambDensity = clamp(ambDensity / clampv, 0.0f, 1.0f);
         if( density + ambDensity > 0.0f )
         {
            totalDensity = density + ambDensity;

            // P = W - colorField.LLC;
            color = valueAtLinearInterpolation4(W, colorField);
            
            // P = W - ambientColorField.LLC;
            amb = valueAtLinearInterpolation4(W, ambientColorField);

            light = gatherLight(W, D, scatterCoefficient, dsmFields, nbLights, phaseFunction, lightColor, lightPosition);

            dT = exp( -scatterCoefficient * ds * totalDensity );
            color = (amb * ambDensity + color * light * density) / totalDensity;
            color = color * T * (1.0f - dT);

            accum += color;
            T *= dT;
         }
         s += ds;
         X += D * ds;
      }
      accum.w = 1.0 - T;
      Cd[item] = accum;
   }
}


// DSM generator

/* Comment out for now; come back later after figuring out host-side
__global__ void ssRayMarchDSMAccumulation(
   float* densityField,
   int densityWidth, int densityHeight, int densityDepth,
   float4 densityLLC, float4 densityRes,
   float* dsmField,
   int dsmWidth, int dsmHeight, int dsmDepth,
   float4 dsmLLC, float4 dsmRes,
   float ds, float4 lightP
)
{
   int4 i;
   i.x = get_global_id(0);
   i.y = get_global_id(1);
   i.z = get_global_id(2);
   i.w = 0;

   float4 tempdist = dsmRes;
   tempdist.x *= dsmWidth;
   tempdist.y *= dsmHeight;
   tempdist.z *= dsmDepth;

   float accum = 0.f;
   float s = 0.f;

   float4 X;
   X.x = (i.x + 0.5f) * dsmRes.x + dsmLLC.x;
   X.y = (i.y + 0.5f) * dsmRes.y + dsmLLC.y;
   X.z = (i.z + 0.5f) * dsmRes.z + dsmLLC.z;
   X.w = 0.f;

   float4 D = lightP - X;
   float maxPathLength = length(D);
   D /= maxPathLength;

   while( s < maxPathLength ){
      float4 Y = X - densityLLC;
      float density = valueAtLinearInterpolation(
         densityField, Y.x, Y.y, Y.z,
         densityRes.x, densityRes.y, densityRes.z,
         densityWidth, densityHeight, densityDepth
      );
      if( density > 0.f && !isnan(density) ){
         accum += density * ds;
      }
      s += ds;
      X += (D * ds);
      Y = X - dsmLLC;

      if( Y.x < 0.f || Y.x > tempdist.x ) { s = maxPathLength; }
      if( Y.y < 0.f || Y.y > tempdist.y ) { s = maxPathLength; }
      if( Y.z < 0.f || Y.z > tempdist.z ) { s = maxPathLength; }
   }

   int index = i.x + dsmWidth * i.y + dsmWidth * dsmHeight * i.z;
   dsmField[index] = accum;
}

__global__ void ssRayMarchDSMWarpAccumulation(
   __read_only image3d_t densityField, float4 densityLLC, float4 densityRes,
   float* dsmField, int dsmWidth, int dsmHeight, int dsmDepth, float4 dsmLLC, float4 dsmRes,
   float ds, float4 lightP,
   __read_only image3d_t warpField, float4 warpLLC, float4 warpRes, int nbWarps
)
{

   int4 i;
   i.x = get_global_id(0);
   i.y = get_global_id(1);
   i.z = get_global_id(2);
   i.w = 0;

   float4 tempdist = dsmRes;
   tempdist.x *= dsmWidth;
   tempdist.y *= dsmHeight;
   tempdist.z *= dsmDepth;
   float maxDistance = length(tempdist);

   float accum = 0.0;
   float s = 0.0;

   float4 X;
   X.x = (i.x + 0.5f) * dsmRes.x + dsmLLC.x;
   X.y = (i.y + 0.5f) * dsmRes.y + dsmLLC.y;
   X.z = (i.z + 0.5f) * dsmRes.z + dsmLLC.z;
   X.w = 0.f;

   float4 D = lightP - X;
   float maxPathLength = length(D);
   D /= maxPathLength;

   while( s < maxPathLength ){
      float4 W;
      float4 Y = X - warpLLC;
      float4 SampY;
      for( int iw = 0; iw < nbWarps; iw++ ){
         W = read_imagef( warpField, nearest, SampY );
         Y -= W;
      }
      Y += warpLLC;
      float density = ( read_imagef(densityField, linear, Y) ).w;
      if( density > 0 && !isnan(density) ){
         accum += density * ds;
      }
      s += ds;
      X += (D * ds);
      Y = X - dsmLLC;
      if( Y.x < 0 || Y.x > tempdist.x ){ s = maxPathLength; }
      if( Y.y < 0 || Y.y > tempdist.y ){ s = maxPathLength; }
      if( Y.z < 0 || Y.z > tempdist.z ){ s = maxPathLength; }
   }

   int index = i.x + dsmWidth * i.y + dsmWidth * dsmHeight * i.z;

   dsmField[index] = accum;
}
*/
