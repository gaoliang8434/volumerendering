#include "hip/hip_runtime.h"

#include "cuRayMarcher.h"

#include "data_interface/include/DataObject.h"
#include "data_interface/include/DataObjectBishop.h"

#include "data_interface/include/ExecutionPolicy.cuh"

#include "cuda/ssRaymarchKernels.cu"

/*
__global__ void ssRayMarchAccumulation(
   const float* densityField, const long* densityMap, const long* densityMapOfMap,
   int densityWidth, int densityHeight, int densityDepth, int densityPartitionSize,
   float4 densityLLC, float4 densityRes, float densityDefVal,

   const float* dsmField, const long* dsmMap, const long* dsmMapOfMap,
   int dsmWidth, int dsmHeight, int dsmDepth, int dsmPartitionSize,
   float4 dsmLLC, float4 dsmRes, float dsmDefVal,

   const float4* colorField, const long* colorMap, const long* colorMapOfMap,
   int colorWidth, int colorHeight, int colorDepth, int colorPartitionSize,
   float4 colorLLC, float4 colorRes, float4 colorDefVal,

   const float4* startPosition, const float4* startDirection, int nbRays,
   float scatterCoefficient, float ds, float maxPathlength,
   float4* Cd
)
*/

using namespace std;

namespace lux
{

void cu_AddDSM( cu_RenderData* d, const ScalarFrustumGrid& dsm )
{
   d->dsmField.push_back( dsm );
}

const ScalarFrustumGrid& cu_GetDSM( cu_RenderData* d, int i )
{
   return d->dsmField[i];
}

void cu_SetDensityField(  cu_RenderData* d, const ScalarGrid& field )
{
    d->densityField = field;
}
void cu_SetAmbientDensityField(  cu_RenderData* d, const ScalarGrid& field )
{
    d->ambientDensityField = field;
}
void cu_SetColorField(  cu_RenderData* d, const ColorGrid& field )
{
    d->colorField = field;
}
void cu_SetAmbientColorField(  cu_RenderData* d, const ColorGrid& field )
{
    d->ambientColorField = field;
}
void cu_AddBoundingBox( cu_RenderData *d, const Vector llc, const Vector urc )
{
   d->boundingBoxes.push_back( AARectangle(llc, urc) );
}

void cu_InitFields( cu_RenderData* d )
{
/*
   // Setup density field struct
   d->density.data = d->densityField->cuData()->getDevicePointer();
   d->density.map = d->densityField->cuMap()->getDevicePointer();
   d->density.mapOfMap = d->densityField->cuMapOfMap()->getDevicePointer();
   d->density.nx = d->densityField->nx();
   d->density.ny = d->densityField->ny();
   d->density.nz = d->densityField->nz();
   d->density.partitionSize = d->densityField->blockSize();

   int psize = d->densityField->blockSize();
   psize *= psize;

   d->density.nnx = d->density.nx / psize;
   d->density.nnx += (d->density.nnx*psize < d->density.nx) ? 1 : 0;

   d->density.nny = d->density.ny / psize;
   d->density.nny += (d->density.nny*psize < d->density.ny) ? 1 : 0;

   d->density.LLC.x = d->densityField->llc()[0];
   d->density.LLC.y = d->densityField->llc()[1];
   d->density.LLC.z = d->densityField->llc()[2];
   d->density.LLC.w = 0.0f;
   d->density.Res.x = d->densityField->dx();
   d->density.Res.y = d->densityField->dy();
   d->density.Res.z = d->densityField->dz();
   d->density.Res.w = 0.0f;
   d->density.defVal = d->densityField->getDefVal();

   // Setup ambient density field struct
   d->ambientDensity.data = d->ambientDensityField->cuData()->getDevicePointer();
   d->ambientDensity.map = d->ambientDensityField->cuMap()->getDevicePointer();
   d->ambientDensity.mapOfMap = d->ambientDensityField->cuMapOfMap()->getDevicePointer();
   d->ambientDensity.nx = d->ambientDensityField->nx();
   d->ambientDensity.ny = d->ambientDensityField->ny();
   d->ambientDensity.nz = d->ambientDensityField->nz();
   d->ambientDensity.partitionSize = d->ambientDensityField->blockSize();

   psize = d->ambientDensityField->blockSize();
   psize *= psize;

   d->ambientDensity.nnx = d->ambientDensity.nx / psize;
   d->ambientDensity.nnx += (d->ambientDensity.nnx*psize < d->ambientDensity.nx) ? 1 : 0;

   d->ambientDensity.nny = d->ambientDensity.ny / psize;
   d->ambientDensity.nny += (d->ambientDensity.nny*psize < d->ambientDensity.ny) ? 1 : 0;

   d->ambientDensity.LLC.x = d->ambientDensityField->llc()[0];
   d->ambientDensity.LLC.y = d->ambientDensityField->llc()[1];
   d->ambientDensity.LLC.z = d->ambientDensityField->llc()[2];
   d->ambientDensity.LLC.w = 0.0f;
   d->ambientDensity.Res.x = d->ambientDensityField->dx();
   d->ambientDensity.Res.y = d->ambientDensityField->dy();
   d->ambientDensity.Res.z = d->ambientDensityField->dz();
   d->ambientDensity.Res.w = 0.0f;
   d->ambientDensity.defVal = d->ambientDensityField->getDefVal();

   // Setup DSM struct array
   d->dsmList.reset( new gilligan::DataObject<cu_DSM>(d->dsmField.size()) );
   cu_DSM* dsm_h = d->dsmList->getHostPointer();
   for(int i = 0; i < d->dsmField.size(); ++i)
   {
      // Camera properties
      float fov = d->dsmField[i]->camera().fov();
      float ar = d->dsmField[i]->camera().aspectRatio();
      Vector eye = d->dsmField[i]->camera().eye();
      Vector view = d->dsmField[i]->camera().view();
      Vector up = d->dsmField[i]->camera().up();
      Vector right = (view ^ up).unitvector();

      dsm_h[i].htanfov = tan(fov * 0.5 * HIP_PI_F / 180.0);
      dsm_h[i].vtanfov = dsm_h[i].htanfov / ar;
      dsm_h[i].near = d->dsmField[i]->camera().nearPlane();
      dsm_h[i].far = d->dsmField[i]->camera().farPlane();

      dsm_h[i].camera_position.x = eye[0];
      dsm_h[i].camera_position.y = eye[1];
      dsm_h[i].camera_position.z = eye[2];
      dsm_h[i].camera_position.w = 0.0f;

      dsm_h[i].camera_right.x = right[0];
      dsm_h[i].camera_right.y = right[1];
      dsm_h[i].camera_right.z = right[2];
      dsm_h[i].camera_right.w = 0.0f;

      dsm_h[i].camera_up.x = up[0];
      dsm_h[i].camera_up.y = up[1];
      dsm_h[i].camera_up.z = up[2];
      dsm_h[i].camera_up.w = 0.0f;

      dsm_h[i].camera_view.x = view[0];
      dsm_h[i].camera_view.y = view[1];
      dsm_h[i].camera_view.z = view[2];
      dsm_h[i].camera_view.w = 0.0f;

      // Field properties
      dsm_h[i].field.data = d->dsmField[i]->cuData()->getDevicePointer();
      dsm_h[i].field.map = d->dsmField[i]->cuMap()->getDevicePointer();
      dsm_h[i].field.mapOfMap = d->dsmField[i]->cuMapOfMap()->getDevicePointer();

      dsm_h[i].field.nx = d->dsmField[i]->nx();
      dsm_h[i].field.ny = d->dsmField[i]->ny();
      dsm_h[i].field.nz = d->dsmField[i]->nz();

      dsm_h[i].field.partitionSize = d->dsmField[i]->blockSize();

      int psize = d->dsmField[i]->blockSize();
      psize *= psize;

      dsm_h[i].field.nnx = dsm_h[i].field.nx / psize;
      dsm_h[i].field.nnx += (dsm_h[i].field.nnx*psize < dsm_h[i].field.nx) ? 1 : 0;

      dsm_h[i].field.nny = dsm_h[i].field.ny / psize;
      dsm_h[i].field.nny += (dsm_h[i].field.nny*psize < dsm_h[i].field.ny) ? 1 : 0;

      Vector llc = d->dsmField[i]->evalP(0, 0, 0);
      dsm_h[i].field.LLC.x = llc[0];
      dsm_h[i].field.LLC.y = llc[1];
      dsm_h[i].field.LLC.z = llc[2];
      dsm_h[i].field.LLC.w = 0.0f;

      dsm_h[i].field.Res.x = d->dsmField[i]->dx();
      dsm_h[i].field.Res.y = d->dsmField[i]->dy();
      dsm_h[i].field.Res.z = d->dsmField[i]->dz();
      dsm_h[i].field.Res.w = 0.0f;

      dsm_h[i].field.defVal = d->dsmField[i]->getDefVal();
   }
   d->dsmList->updateDevice();

   // Setup color field struct
   d->color.data = d->colorField->cuData()->getDevicePointer();
   d->color.map = d->colorField->cuMap()->getDevicePointer();
   d->color.mapOfMap = d->colorField->cuMapOfMap()->getDevicePointer();
   d->color.nx = d->colorField->nx();
   d->color.ny = d->colorField->ny();
   d->color.nz = d->colorField->nz();
   d->color.partitionSize = d->colorField->blockSize();

   psize = d->colorField->blockSize();
   psize *= psize;

   d->color.nnx = d->color.nx / psize;
   d->color.nnx += (d->color.nnx*psize < d->color.nx) ? 1 : 0;

   d->color.nny = d->color.ny / psize;
   d->color.nny += (d->color.nny*psize < d->color.ny) ? 1 : 0;

   d->color.LLC.x = d->colorField->llc()[0];
   d->color.LLC.y = d->colorField->llc()[1];
   d->color.LLC.z = d->colorField->llc()[2];
   d->color.LLC.w = 0.0f;
   d->color.Res.x = d->colorField->dx();
   d->color.Res.y = d->colorField->dy();
   d->color.Res.z = d->colorField->dz();
   d->color.Res.w = 0.0f;
   d->color.defVal.x = d->colorField->getDefVal()[0];
   d->color.defVal.y = d->colorField->getDefVal()[1];
   d->color.defVal.z = d->colorField->getDefVal()[2];
   d->color.defVal.w = d->colorField->getDefVal()[3];

   // Setup color field struct
   d->ambientColor.data = d->ambientColorField->cuData()->getDevicePointer();
   d->ambientColor.map = d->ambientColorField->cuMap()->getDevicePointer();
   d->ambientColor.mapOfMap = d->ambientColorField->cuMapOfMap()->getDevicePointer();
   d->ambientColor.nx = d->ambientColorField->nx();
   d->ambientColor.ny = d->ambientColorField->ny();
   d->ambientColor.nz = d->ambientColorField->nz();
   d->ambientColor.partitionSize = d->ambientColorField->blockSize();

   psize = d->ambientColorField->blockSize();
   psize *= psize;

   d->ambientColor.nnx = d->ambientColor.nx / psize;
   d->ambientColor.nnx += (d->ambientColor.nnx*psize < d->ambientColor.nx) ? 1 : 0;

   d->ambientColor.nny = d->ambientColor.ny / psize;
   d->ambientColor.nny += (d->ambientColor.nny*psize < d->ambientColor.ny) ? 1 : 0;

   d->ambientColor.LLC.x = d->ambientColorField->llc()[0];
   d->ambientColor.LLC.y = d->ambientColorField->llc()[1];
   d->ambientColor.LLC.z = d->ambientColorField->llc()[2];
   d->ambientColor.LLC.w = 0.0f;
   d->ambientColor.Res.x = d->ambientColorField->dx();
   d->ambientColor.Res.y = d->ambientColorField->dy();
   d->ambientColor.Res.z = d->ambientColorField->dz();
   d->ambientColor.Res.w = 0.0f;
   d->ambientColor.defVal.x = d->ambientColorField->getDefVal()[0];
   d->ambientColor.defVal.y = d->ambientColorField->getDefVal()[1];
   d->ambientColor.defVal.z = d->ambientColorField->getDefVal()[2];
   d->ambientColor.defVal.w = d->ambientColorField->getDefVal()[3];
*/
}

void cu_ssRayMarchAccumulation( const cu_RenderData& input, vector<Color>& Cd )
{
   int psize;

   Cd.clear();
   Cd.resize( input.startPosition.size() );

   /* Allocate device memory for std::vector<T>s:
    * Cd
    * startPosition
    * startDirection
    * lightColor
    * lightPosition
    */

   gilligan::DataObject<Color> cu_Cd(Cd.size());
   for(int i = 0; i < Cd.size(); ++i)
   {
      (*cu_Cd)[i].x = Cd[i][0];
      (*cu_Cd)[i].y = Cd[i][1];
      (*cu_Cd)[i].z = Cd[i][2];
      (*cu_Cd)[i].w = Cd[i][3];
   }
   cu_Cd.updateDevice();

   gilligan::DataObject<Vector> cu_StartPosition(input.startPosition.size());
   for(int i = 0; i < input.startPosition.size(); ++i)
   {
      (*cu_StartPosition)[i].x = input.startPosition[i][0];
      (*cu_StartPosition)[i].y = input.startPosition[i][1];
      (*cu_StartPosition)[i].z = input.startPosition[i][2];
      (*cu_StartPosition)[i].w = 0.0f;
   }
   cu_StartPosition.updateDevice();

   gilligan::DataObject<Vector> cu_StartDirection(input.startDirection.size());
   for(int i = 0; i < input.startDirection.size(); ++i)
   {
      (*cu_StartDirection)[i].x = input.startDirection[i][0];
      (*cu_StartDirection)[i].y = input.startDirection[i][1];
      (*cu_StartDirection)[i].z = input.startDirection[i][2];
      (*cu_StartDirection)[i].w = 0.0f;
   }
   cu_StartDirection.updateDevice();

   gilligan::DataObject<Color> cu_LightColor(input.lightColor.size());
   for(int i = 0; i < input.lightColor.size(); ++i)
   {
      (*cu_LightColor)[i].x = input.lightColor[i][0];
      (*cu_LightColor)[i].y = input.lightColor[i][1];
      (*cu_LightColor)[i].z = input.lightColor[i][2];
      (*cu_LightColor)[i].w = input.lightColor[i][3];
   }
   cu_LightColor.updateDevice();

   gilligan::DataObject<Vector> cu_LightPosition(input.lightPosition.size());
   for(int i = 0; i < input.lightPosition.size(); ++i)
   {
      (*cu_LightPosition)[i].x = input.lightPosition[i][0];
      (*cu_LightPosition)[i].y = input.lightPosition[i][1];
      (*cu_LightPosition)[i].z = input.lightPosition[i][2];
      (*cu_LightPosition)[i].w = 0.0f;
   }
   cu_LightPosition.updateDevice();

   // Setup density field struct
   cu_ScalarField density;
   density.data = input.densityField->cuData()->getDevicePointer();
   density.map = input.densityField->cuMap()->getDevicePointer();
   density.mapOfMap = input.densityField->cuMapOfMap()->getDevicePointer();
   density.nx = input.densityField->nx();
   density.ny = input.densityField->ny();
   density.nz = input.densityField->nz();
   density.partitionSize = input.densityField->blockSize();

   psize = input.densityField->blockSize();
   psize *= psize;

   density.nnx = density.nx / psize;
   density.nnx += (density.nnx*psize < density.nx) ? 1 : 0;

   density.nny = density.ny / psize;
   density.nny += (density.nny*psize < density.ny) ? 1 : 0;

   density.LLC.x = input.densityField->llc()[0];
   density.LLC.y = input.densityField->llc()[1];
   density.LLC.z = input.densityField->llc()[2];
   density.LLC.w = 0.0f;
   density.Res.x = input.densityField->dx();
   density.Res.y = input.densityField->dy();
   density.Res.z = input.densityField->dz();
   density.Res.w = 0.0f;
   density.defVal = input.densityField->getDefVal();

   // Setup ambient density field struct
   cu_ScalarField ambientDensity;
   ambientDensity.data = input.ambientDensityField->cuData()->getDevicePointer();
   ambientDensity.map = input.ambientDensityField->cuMap()->getDevicePointer();
   ambientDensity.mapOfMap = input.ambientDensityField->cuMapOfMap()->getDevicePointer();
   ambientDensity.nx = input.ambientDensityField->nx();
   ambientDensity.ny = input.ambientDensityField->ny();
   ambientDensity.nz = input.ambientDensityField->nz();
   ambientDensity.partitionSize = input.ambientDensityField->blockSize();

   psize = input.ambientDensityField->blockSize();
   psize *= psize;

   ambientDensity.nnx = ambientDensity.nx / psize;
   ambientDensity.nnx += (ambientDensity.nnx*psize < ambientDensity.nx) ? 1 : 0;

   ambientDensity.nny = ambientDensity.ny / psize;
   ambientDensity.nny += (ambientDensity.nny*psize < ambientDensity.ny) ? 1 : 0;

   ambientDensity.LLC.x = input.ambientDensityField->llc()[0];
   ambientDensity.LLC.y = input.ambientDensityField->llc()[1];
   ambientDensity.LLC.z = input.ambientDensityField->llc()[2];
   ambientDensity.LLC.w = 0.0f;
   ambientDensity.Res.x = input.ambientDensityField->dx();
   ambientDensity.Res.y = input.ambientDensityField->dy();
   ambientDensity.Res.z = input.ambientDensityField->dz();
   ambientDensity.Res.w = 0.0f;
   ambientDensity.defVal = input.ambientDensityField->getDefVal();

   // Setup DSM struct array
   gilligan::DataObject<cu_DSM> dsmList(input.dsmField.size());
   cu_DSM* dsm_h = dsmList.getHostPointer();
   for(int i = 0; i < input.dsmField.size(); ++i)
   {
      // Camera properties
      float fov = input.dsmField[i]->camera().fov();
      float ar = input.dsmField[i]->camera().aspectRatio();
      Vector eye = input.dsmField[i]->camera().eye();
      Vector view = input.dsmField[i]->camera().view();
      Vector up = input.dsmField[i]->camera().up();
      Vector right = (view ^ up).unitvector();

      dsm_h[i].htanfov = tan(fov * 0.5 * HIP_PI_F / 180.0);
      dsm_h[i].vtanfov = dsm_h[i].htanfov / ar;
      dsm_h[i].near = input.dsmField[i]->camera().nearPlane();
      dsm_h[i].far = input.dsmField[i]->camera().farPlane();

      dsm_h[i].camera_position.x = eye[0];
      dsm_h[i].camera_position.y = eye[1];
      dsm_h[i].camera_position.z = eye[2];
      dsm_h[i].camera_position.w = 0.0f;

      dsm_h[i].camera_right.x = right[0];
      dsm_h[i].camera_right.y = right[1];
      dsm_h[i].camera_right.z = right[2];
      dsm_h[i].camera_right.w = 0.0f;

      dsm_h[i].camera_up.x = up[0];
      dsm_h[i].camera_up.y = up[1];
      dsm_h[i].camera_up.z = up[2];
      dsm_h[i].camera_up.w = 0.0f;

      dsm_h[i].camera_view.x = view[0];
      dsm_h[i].camera_view.y = view[1];
      dsm_h[i].camera_view.z = view[2];
      dsm_h[i].camera_view.w = 0.0f;

      // Field properties
      dsm_h[i].field.data = input.dsmField[i]->cuData()->getDevicePointer();
      dsm_h[i].field.map = input.dsmField[i]->cuMap()->getDevicePointer();
      dsm_h[i].field.mapOfMap = input.dsmField[i]->cuMapOfMap()->getDevicePointer();

      dsm_h[i].field.nx = input.dsmField[i]->nx();
      dsm_h[i].field.ny = input.dsmField[i]->ny();
      dsm_h[i].field.nz = input.dsmField[i]->nz();

      dsm_h[i].field.partitionSize = input.dsmField[i]->blockSize();

      int psize = input.dsmField[i]->blockSize();
      psize *= psize;

      dsm_h[i].field.nnx = dsm_h[i].field.nx / psize;
      dsm_h[i].field.nnx += (dsm_h[i].field.nnx*psize < dsm_h[i].field.nx) ? 1 : 0;

      dsm_h[i].field.nny = dsm_h[i].field.ny / psize;
      dsm_h[i].field.nny += (dsm_h[i].field.nny*psize < dsm_h[i].field.ny) ? 1 : 0;

      Vector llc = input.dsmField[i]->evalP(0, 0, 0);
      dsm_h[i].field.LLC.x = llc[0];
      dsm_h[i].field.LLC.y = llc[1];
      dsm_h[i].field.LLC.z = llc[2];
      dsm_h[i].field.LLC.w = 0.0f;

      dsm_h[i].field.Res.x = input.dsmField[i]->dx();
      dsm_h[i].field.Res.y = input.dsmField[i]->dy();
      dsm_h[i].field.Res.z = input.dsmField[i]->dz();
      dsm_h[i].field.Res.w = 0.0f;

      dsm_h[i].field.defVal = input.dsmField[i]->getDefVal();
   }
   dsmList.updateDevice();

   // Setup color field struct
   cu_ColorField color;
   color.data = input.colorField->cuData()->getDevicePointer();
   color.map = input.colorField->cuMap()->getDevicePointer();
   color.mapOfMap = input.colorField->cuMapOfMap()->getDevicePointer();
   color.nx = input.colorField->nx();
   color.ny = input.colorField->ny();
   color.nz = input.colorField->nz();
   color.partitionSize = input.colorField->blockSize();

   psize = input.colorField->blockSize();
   psize *= psize;

   color.nnx = color.nx / psize;
   color.nnx += (color.nnx*psize < color.nx) ? 1 : 0;

   color.nny = color.ny / psize;
   color.nny += (color.nny*psize < color.ny) ? 1 : 0;

   color.LLC.x = input.colorField->llc()[0];
   color.LLC.y = input.colorField->llc()[1];
   color.LLC.z = input.colorField->llc()[2];
   color.LLC.w = 0.0f;
   color.Res.x = input.colorField->dx();
   color.Res.y = input.colorField->dy();
   color.Res.z = input.colorField->dz();
   color.Res.w = 0.0f;
   color.defVal.x = input.colorField->getDefVal()[0];
   color.defVal.y = input.colorField->getDefVal()[1];
   color.defVal.z = input.colorField->getDefVal()[2];
   color.defVal.w = input.colorField->getDefVal()[3];

   // Setup color field struct
   cu_ColorField ambientColor;
   ambientColor.data = input.ambientColorField->cuData()->getDevicePointer();
   ambientColor.map = input.ambientColorField->cuMap()->getDevicePointer();
   ambientColor.mapOfMap = input.ambientColorField->cuMapOfMap()->getDevicePointer();
   ambientColor.nx = input.ambientColorField->nx();
   ambientColor.ny = input.ambientColorField->ny();
   ambientColor.nz = input.ambientColorField->nz();
   ambientColor.partitionSize = input.ambientColorField->blockSize();

   psize = input.ambientColorField->blockSize();
   psize *= psize;

   ambientColor.nnx = ambientColor.nx / psize;
   ambientColor.nnx += (ambientColor.nnx*psize < ambientColor.nx) ? 1 : 0;

   ambientColor.nny = ambientColor.ny / psize;
   ambientColor.nny += (ambientColor.nny*psize < ambientColor.ny) ? 1 : 0;

   ambientColor.LLC.x = input.ambientColorField->llc()[0];
   ambientColor.LLC.y = input.ambientColorField->llc()[1];
   ambientColor.LLC.z = input.ambientColorField->llc()[2];
   ambientColor.LLC.w = 0.0f;
   ambientColor.Res.x = input.ambientColorField->dx();
   ambientColor.Res.y = input.ambientColorField->dy();
   ambientColor.Res.z = input.ambientColorField->dz();
   ambientColor.Res.w = 0.0f;
   ambientColor.defVal.x = input.ambientColorField->getDefVal()[0];
   ambientColor.defVal.y = input.ambientColorField->getDefVal()[1];
   ambientColor.defVal.z = input.ambientColorField->getDefVal()[2];
   ambientColor.defVal.w = input.ambientColorField->getDefVal()[3];

   size_t threads_per_block = 256;
   size_t grid_size = input.startPosition.size() / threads_per_block;
   if( grid_size * threads_per_block < input.startPosition.size() ) grid_size++;
   gilligan::util::ExecutionPolicy policy(threads_per_block, grid_size);

   int nbRays = Cd.size();

   // Launch CUDA kernel with ExecutionPolicy
   kernel_ssRayMarchAccumulation<<< policy.gridSize(), policy.blockSize() >>>(
      density, dsmList.getDevicePointer(), color, ambientDensity, ambientColor,

      cu_LightColor.getDevicePointer(), cu_LightPosition.getDevicePointer(), input.dsmField.size(),

      cu_StartPosition.getDevicePointer(), cu_StartDirection.getDevicePointer(), nbRays,
      input.scatterCoefficient, input.ds, input.maxPathlength, input.clampv, input.phaseF,

      cu_Cd.getDevicePointer()
   );

   hipDeviceSynchronize();
   hipError_t res = hipPeekAtLastError();
   if (res != hipSuccess)
   {
      std::cout << "CUDA Error: " << hipGetErrorString(res) << std::endl;
   }

   // Transfer back cu_Cd from device
   cu_Cd.updateHost();
   for(int i = 0; i < Cd.size(); ++i)
   {
      Cd[i][0] = (*cu_Cd)[i].x;
      Cd[i][1] = (*cu_Cd)[i].y;
      Cd[i][2] = (*cu_Cd)[i].z;
      Cd[i][3] = (*cu_Cd)[i].w;
   }
}


void cu_SetUniformPhaseFunction( cu_RenderData *d, float value )
{
   d->phaseF.type = cuPhaseFunction::PhaseFunctionType::UNIFORM;
   d->phaseF.val = value;
}

void cu_SetHenyeyGreensteinPhaseFunction( cu_RenderData *d, float g )
{
   d->phaseF.type = cuPhaseFunction::PhaseFunctionType::HENYEYGREENSTEIN;
   d->phaseF.g0 = g;
}

void cu_SetDoubleHenyeyGreensteinPhaseFunction( cu_RenderData *d, float g0, float g1, float mix )
{
   d->phaseF.type = cuPhaseFunction::PhaseFunctionType::DOUBLE_HENYEYGREENSTEIN;
   d->phaseF.g0 = g0;
   d->phaseF.g1 = g1;
   d->phaseF.mix = mix;
}

void cu_SetFournierForandPhaseFunction( cu_RenderData *d, float en, float mu )
{
   d->phaseF.type = cuPhaseFunction::PhaseFunctionType::FOURNIERFORAND;
   d->phaseF.en = en;
   d->phaseF.mu = mu;

   d->phaseF.nu = (3.0 - mu) / 2.0;
   d->phaseF.delta180 = 4.0 / ( 3.0 * (en-1.0)*(en-1.0) );
   double delta180nupower = std::pow( d->phaseF.delta180, d->phaseF.nu );
   d->phaseF.p1factor =  (1.0 - delta180nupower) / ( 16.0 * M_PI * delta180nupower * (d->phaseF.delta180-1.0) );
}

}
